#include "hip/hip_runtime.h"
//
// Created by Guannan Guo on 5/27/20.
//

#include "CUDACalculator.h"
#include <iostream>

// define hardware dependent variables
static const int blockSize = 1024;
static const int gridSize = 24; //this number is hardware-dependent; usually #SM*2 is a good number.

__global__ void sumCommMultiBlock(const double *gArr, int arraySize, double *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    double sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

double sumArray(double* roi_pointer, int arraySize) {
//    int* roi_in;
//    hipMalloc(&roi_in, arraySize);
//    hipMemcpy(roi_in, roi_domain.data, arraySize, hipMemcpyHostToDevice);
    // setting cache and shared modes
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    // transfer data from host to device
    double* img_in;
    hipMalloc(&img_in, arraySize * sizeof(double));
    hipMemcpy(img_in, roi_pointer, arraySize * sizeof(double), hipMemcpyHostToDevice);

    double roi_sum;
    double* img_out;
    hipMalloc((void**)&img_out, sizeof(double)*gridSize);
    // call the sum function
    sumCommMultiBlock<<<gridSize, blockSize>>>(img_in, arraySize, img_out);
    sumCommMultiBlock<<<1, blockSize>>>(img_out, gridSize, img_out);
    hipDeviceSynchronize();
    // transfer data from device to host
    hipMemcpy(&roi_sum, img_out, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(img_in);
    hipFree(img_out);
    return roi_sum;
}