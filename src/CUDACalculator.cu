#include "hip/hip_runtime.h"
//
// Created by Guannan Guo on 5/27/20.
//

#include "CUDACalculator.h"

__global__ void cuda_calculate_greyscale_kernel(const hipTextureObject_t tex_img, const int pixels, int* const res) {
    const unsigned int x = (blockIdx.x << 8) + threadIdx.x;
    const uint8_t sub_res = tex1Dfetch<int>(tex_img, 3 * x) + tex1Dfetch<int>(tex_img, 3 * x + 1) + tex1Dfetch<int>(tex_img, 3 * x + 2);
    if (x < pixels) {
        res[0] = res[0] + sub_res;
    }
}

void cuda_calculate_greyscale(const hipTextureObject_t tex_img, const int pixels, int* const res) {
    cuda_calculate_greyscale_kernel<<<((pixels - 1) >> 8) + 1, 256>>>(tex_img, pixels, res);
    hipDeviceSynchronize();
}

