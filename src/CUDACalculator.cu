#include "hip/hip_runtime.h"
//
// Created by Guannan Guo on 5/27/20.
//

#include "CUDACalculator.h"

// define hardware dependent variables
static const int blockSize = 1024;
static const int gridSize = 24; //this number is hardware-dependent; usually #SM*2 is a good number.

__global__ void sumCommMultiBlock(const long int *gArr, int arraySize, long int *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    long int sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ long int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

long int sumArray(long int* roi_pointer, int arraySize) {
//    int* roi_in;
//    hipMalloc(&roi_in, arraySize);
//    hipMemcpy(roi_in, roi_domain.data, arraySize, hipMemcpyHostToDevice);
    // setting cache and shared modes
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    // transfer data from host to device
    long int* img_in;
    hipMalloc(&img_in, arraySize * sizeof(long int));
    hipMemcpy(img_in, roi_pointer, arraySize * sizeof(long int), hipMemcpyHostToDevice);

    long int roi_sum;
    long int* img_out;
    hipMalloc((void**)&img_out, sizeof(long int)*gridSize);
    // call the sum function
    sumCommMultiBlock<<<gridSize, blockSize>>>(img_in, arraySize, img_out);
    sumCommMultiBlock<<<1, blockSize>>>(img_out, gridSize, img_out);
    hipDeviceSynchronize();
    // transfer data from device to host
    hipMemcpy(&roi_sum, img_out, sizeof(long int), hipMemcpyDeviceToHost);
    hipFree(img_in);
    hipFree(img_out);
    return roi_sum;
}