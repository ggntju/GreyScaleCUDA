#include "hip/hip_runtime.h"
//
// Created by Guannan Guo on 5/27/20.
//

#include "CUDACalculator.h"

// define hardware dependent variables
static const int blockSize = 1024;
static const int gridSize = 12; //this number is hardware-dependent; usually #SM*2 is a good number.

__global__ void sumCommMultiBlock(const int *gArr, int arraySize, int *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

int sumArray(Mat roi_in) {
    int arraySize = 3 * roi_in.rows * roi_in.cols;
    // setting cache and shared modes
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    // transfer data from host to device
    int* img_in;
    hipMalloc(&img_in, arraySize);
    hipMemcpy(img_in, roi_in.data, arraySize, hipMemcpyHostToDevice);

    int roi_sum;
    int* img_out;
    hipMalloc((void**)&img_out, sizeof(int)*gridSize);
    // call the sum function
    sumCommMultiBlock<<<gridSize, blockSize>>>(img_in, arraySize, img_out);
    sumCommMultiBlock<<<1, blockSize>>>(img_out, gridSize, img_out);
    hipDeviceSynchronize();
    // transfer data from device to host
    hipMemcpy(&roi_sum, img_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(img_in);
    hipFree(img_out);
    return roi_sum;
}